#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#include <torch/torch.h>
#include <hipblas.h>
#include <ATen/cuda/CUDABlas.h>
#include <torch/extension.h>
#include <iostream>
using namespace std;

#include <iostream>
using namespace std;

__global__ void GEMMLowpKernel(const float* in, const int N, float* out,
                               float scale, float shift, long long qmax, const float* noise, bool enforce_true_zero) {
//   printf("Potting Success\n");
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
      out[i] = in[i];
      if (enforce_true_zero)
        out[i] = (out[i] / scale) + shift;
      else
        out[i] = (out[i] + shift) / scale;
      out[i] += noise[i];
      out[i] = fminf(out[i], qmax);
      out[i] = fmaxf(out[i], 0.);
      out[i] = roundf(out[i]);
      if (enforce_true_zero)
        out[i] = (out[i] - shift) * scale;
      else
        out[i] = out[i] * scale - shift;
  }
}


#define block_count 32
#define thread_per_block 1024
// Wrapper for ATen
at::Tensor float2gemmlowp(at::Tensor in, float range, float offset, int num_bits, bool int_exp, bool enforce_true_zero, at::Tensor noise) {
    if (range <= 0)
        return in;

    int N = in.numel();
    auto out = at::zeros_like(in);
    long long qmax = (0x1l << num_bits) - 1;
    float scale = range / qmax;
    if (int_exp)
        scale = powf(2, int(ceilf(log2f(scale))));
    float zero_point = roundf(-offset / scale);
    float shift = enforce_true_zero ? zero_point : -offset;
    GEMMLowpKernel<<<block_count, thread_per_block>>>(in.data<float>(), N, out.data<float>(), scale, shift, qmax, noise.data<float>(), enforce_true_zero);

    return out;
}

void cublasGemm(int TA, int TB, int M, int N, int K, float ALPHA,
                at::Tensor A_gpu, int lda,
                at::Tensor B_gpu, int ldb,
                float BETA,
                at::Tensor C_gpu, int ldc)
{
    //cout << A_gpu.type() << endl;
    //cout << B_gpu.type() << endl;
    //cout << C_gpu.type() << endl;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    //printf("some : %d\n", CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    /*
    hipError_t status = static_cast<hipError_t>(
                                                    hipblasGemmEx(
                                                    handle,
                                                    (TA ? HIPBLAS_OP_T : HIPBLAS_OP_N),
                                                    (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N),
                                                    M, N, K,
                                                    &ALPHA,
                                                    A_gpu.data_ptr<int8_t>(), HIP_R_8I, lda,
                                                    B_gpu.data_ptr<int8_t>(), HIP_R_8I, ldb,
                                                    &BETA,
                                                    C_gpu.data_ptr<int32_t>(), HIP_R_32I, ldc,
                                                    HIPBLAS_COMPUTE_32I,
                                                    A_gpu.data_ptr<float>(), HIP_R_32F, lda,
                                                    B_gpu.data_ptr<float>(), HIP_R_32F, ldb,
                                                    &BETA,
                                                    C_gpu.data_ptr<float>(), HIP_R_32F, ldc,
                                                    HIP_R_32F,
                                                    CUBLAS_GEMM_DEFAULT_TENSOR_OP)
                                                );
//     hipblasStatus_t status = hipblasGemmEx(
//                                                 handle,
//                                                 (TA ? HIPBLAS_OP_T : HIPBLAS_OP_N),
//                                                 (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N),
//                                                 M, N, K,
//                                                 &ALPHA,
//                                                 A_gpu.data_ptr<float>(), HIP_R_32F, lda,
//                                                 B_gpu.data_ptr<float>(), HIP_R_32F, ldb,
//                                                 &BETA,
//                                                 C_gpu.data_ptr<float>(), HIP_R_32F, ldc,
//                                                 HIP_R_32F,
//                                                 CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    */
    hipblasStatus_t status = hipblasGemmEx(
                                                handle,
                                                (TA ? HIPBLAS_OP_T : HIPBLAS_OP_N),
                                                (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N),
                                                M, N, K,
                                                &ALPHA,
                                                A_gpu.data_ptr<int8_t>(), HIP_R_8I, lda,
                                                B_gpu.data_ptr<int8_t>(), HIP_R_8I, ldb,
                                                &BETA,
                                                C_gpu.data_ptr<float>(), HIP_R_32F, ldc,
                                                HIPBLAS_COMPUTE_32F,
                                                CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    //if (status == HIPBLAS_STATUS_EXECUTION_FAILED) printf("start 1 Found");
    //if (status == HIPBLAS_STATUS_INTERNAL_ERROR) printf("Found");
    //printf("%d\n", status);
//     printf("%s\n", hipGetErrorName(status));
}
