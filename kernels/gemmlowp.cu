#include "hip/hip_runtime.h"
#include <vector>

#include <torch/torch.h>
#include <hipblaslt.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <ATen/ATen.h>
#include <ATen/cuda/CUDABlas.h>
#include <torch/extension.h>


#include <iostream>
using namespace std;

__global__ void GEMMLowpKernel(const float* in, const int N, float* out,
                               float scale, float shift, long long qmax, const float* noise, bool enforce_true_zero) {
//   printf("Potting Success\n");
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
      out[i] = in[i];
      if (enforce_true_zero)
        out[i] = (out[i] / scale) + shift;
      else
        out[i] = (out[i] + shift) / scale;
      out[i] += noise[i];
      out[i] = fminf(out[i], qmax);
      out[i] = fmaxf(out[i], 0.);
      out[i] = roundf(out[i]);
      if (enforce_true_zero)
        out[i] = (out[i] - shift) * scale;
      else
        out[i] = out[i] * scale - shift;
  }
}


#define block_count 32
#define thread_per_block 1024
// Wrapper for ATen
at::Tensor float2gemmlowp(at::Tensor in, float range, float offset, int num_bits, bool int_exp, bool enforce_true_zero, at::Tensor noise) {
    if (range <= 0)
        return in;

    int N = in.numel();
    auto out = at::zeros_like(in);
    long long qmax = (0x1l << num_bits) - 1;
    float scale = range / qmax;
    if (int_exp)
        scale = powf(2, int(ceilf(log2f(scale))));
    float zero_point = roundf(-offset / scale);
    float shift = enforce_true_zero ? zero_point : -offset;
    GEMMLowpKernel<<<block_count, thread_per_block>>>(in.data<float>(), N, out.data<float>(), scale, shift, qmax, noise.data<float>(), enforce_true_zero);

    return out;
}

//hipblasLtHandle_t ltHandle
void cublasLtGemm(int TA, int TB,
                    int m, int n, int k,
                    const float *ALPHA, 
                    at::Tensor A_gpu, int lda,
                    at::Tensor B_gpu, int ldb,
                    const float *BETA,
                    at::Tensor C_gpu, int ldc,
                    void *workspace,
                    size_t workspaceSize)
{
    hipblasLtHandle_t ltHandle;
    hipblasLtCreate(&ltHandle);
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

    hipblasLtMatmulDesc_t opDesc = NULL;
    hipblasLtMatrixLayout_t Adesc=NULL, Bdesc=NULL, Cdesc=NULL;
    //hipblasLtMatmulPreference_t preference = NULL;

    //int returnedResults = 0;
    //hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    hipblasOperation_t transa = (TA ? HIPBLAS_OP_T : HIPBLAS_OP_N);
    hipblasOperation_t transb = (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N);
    status = hipblasLtMatmulDescCreate(&opDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
    status = hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
    status = hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb));

    status = hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8I, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
    status = hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8I, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
    //status = hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8I, k, k, lda);
    //status = hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8I, m, k, ldb);
    status = hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, m, n, ldc);

    //status = hipblasLtMatmulPreferenceCreate(&preference);
    //status = hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));

    hipblasStatus_t stat = hipblasLtMatmul(
                                            ltHandle,
                                            opDesc,
                                            ALPHA,
                                            A_gpu.data_ptr<int8_t>(), Adesc,  // Input
                                            B_gpu.data_ptr<int8_t>(), Bdesc,  // Input
                                            BETA,
                                            C_gpu.data_ptr(), Cdesc, // Bias
                                            C_gpu.data_ptr<float>(), Cdesc,// Output
                                            NULL,
                                            workspace,
                                            workspaceSize,
                                            0);

}

void cublasGemm(int TA, int TB, int M, int N, int K, float ALPHA,
                torch::Tensor A_gpu, int lda,
                torch::Tensor B_gpu, int ldb,
                float BETA,
                torch::Tensor C_gpu, int ldc)
{
    //cout << A_gpu.type() << endl;
    //cout << B_gpu.type() << endl;
    //cout << C_gpu.type() << endl;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
    /*
    hipError_t status = static_cast<hipError_t>(
                                                    hipblasGemmEx(
                                                    handle,
                                                    (TA ? HIPBLAS_OP_T : HIPBLAS_OP_N),
                                                    (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N),
                                                    M, N, K,
                                                    &ALPHA,
                                                    A_gpu.data_ptr<int8_t>(), HIP_R_8I, lda,
                                                    B_gpu.data_ptr<int8_t>(), HIP_R_8I, ldb,

                                                    &BETA,
                                                    C_gpu.data_ptr<int32_t>(), HIP_R_32I, ldc,
                                                    HIPBLAS_COMPUTE_32I,
                                                    A_gpu.data_ptr<float>(), HIP_R_32F, lda,
                                                    B_gpu.data_ptr<float>(), HIP_R_32F, ldb,
                                                    &BETA,
                                                    C_gpu.data_ptr<int32_t>(), HIP_R_32I, ldc,
                                                    HIPBLAS_COMPUTE_32I,
                                                    CUBLAS_GEMM_DEFAULT_TENSOR_OP)
                                                );
//     hipError_t status = static_cast<hipError_t>(
//                                                     hipblasGemmEx(
//                                                     handle,
//                                                     (TA ? HIPBLAS_OP_T : HIPBLAS_OP_N),
//                                                     (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N),
//                                                     M, N, K,
//                                                     &ALPHA,
//                                                     A_gpu.data_ptr<float>(), HIP_R_32F, lda,
//                                                     B_gpu.data_ptr<float>(), HIP_R_32F, ldb,
//                                                     &BETA,
//                                                     C_gpu.data_ptr<float>(), HIP_R_32F, ldc,
//                                                     HIP_R_32F,
//                                                     CUBLAS_GEMM_DEFAULT_TENSOR_OP)
//                                                 );
//     hipblasStatus_t status = hipblasGemmEx(
//                                                 handle,
//                                                 (TA ? HIPBLAS_OP_T : HIPBLAS_OP_N),
//                                                 (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N),
//                                                 M, N, K,
//                                                 &ALPHA,
//                                                 A_gpu.data_ptr<float>(), HIP_R_32F, lda,
//                                                 B_gpu.data_ptr<float>(), HIP_R_32F, ldb,
//                                                 &BETA,
//                                                 C_gpu.data_ptr<float>(), HIP_R_32F, ldc,
//                                                 HIP_R_32F,
//                                                 CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    */
    // Cublas Gemm
    hipblasStatus_t status = hipblasGemmEx(
                                                handle,
                                                (TA ? HIPBLAS_OP_T : HIPBLAS_OP_N),
                                                (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N),
                                                M, N, K,
                                                static_cast<const void*>(&ALPHA),
                                                static_cast<const void*>(A_gpu.data_ptr()), HIP_R_16F, lda,
                                                static_cast<const void*>(B_gpu.data_ptr()), HIP_R_16F, ldb,
                                                static_cast<const void*>(&BETA),
                                                static_cast<void*>(C_gpu.data_ptr()), HIP_R_16F, ldc,
                                                HIPBLAS_COMPUTE_16F,
                                                CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    // CUBLAS_GEMM_DEFAULT_TENSOR_OP
    //if (status == HIPBLAS_STATUS_EXECUTION_FAILED) printf("start 1 Found");
    //if (status == HIPBLAS_STATUS_INTERNAL_ERROR) printf("Found");
    //printf("%d\n", status);
//     printf("%s\n", hipGetErrorName(status));
}
